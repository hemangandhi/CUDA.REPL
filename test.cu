#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void origin ( int * n, int k ){
int thid = threadIdx.x + blockDim.x * blockIdx.x;
if (thid >= k) return;
n[thid] = thid;
}
int main(){
int n[] = {2,2,2,2};
int * gpu_n;
hipMalloc( &gpu_n, sizeof(int) * 4);
hipMemcpy(gpu_n, n, sizeof(int) * 4, hipMemcpyHostToDevice);
origin<<<2,2>>>( gpu_n,4);
printf("Last error: %s \n", hipGetErrorString(hipDeviceSynchronize()));
hipMemcpy(n, gpu_n, sizeof(int) * 4, hipMemcpyDeviceToHost);
hipFree( gpu_n);
for(int i = 0; i < 4; i++)
printf("n[%d] : %d\n", i, n[i]);
}