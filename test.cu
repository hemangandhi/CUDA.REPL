#include "hip/hip_runtime.h"
#include "cudaRuntime.h"
__global__ void default (  int * wat  ){
int thid = threadIdx.x + blockDim.x * blockIdx.x;

wat[thid] = thid;

}
int main(){
int wat[] = {};
int * gpu_wat;
hipMalloc( &gpu_wat, sizeof(int) * 1);
hipMemcpy(gpu_wat, wat, sizeof(int) * 1, hipMemcpyHostToDevice);
default<<<,>>>( gpu_,);
printf("Last error: %s \n", hipGetErrorString(hipDeviceSynchronize()));
hipMemcpy(wat, gpu_wat, sizeof(int) * 1, hipMemcpyDeviceToHost);
hipFree( gpu_wat);
for(int i = 0; i < 1; i++)
printf("wat[%d] : \n", i, wat[i]);
}